#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include ""
#define LOG_INPUT if(0)
#define LOG_OUTPUT if(1)
#define LOG if(0)


__global__ void matrix_add(float* A, float* B, float* C, int M, int N)
{
    // Complete the kernel code snippet
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
void vecAdd(float* A, float* B, float* C, int M, int N)
{
    //host program
    int size = N * sizeof(float);
    float* d_A = NULL, * d_B = NULL, * d_C = NULL;
    hipError_t err = hipSuccess;
}

void print_matrix(float* A, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
            printf("%.2f ", A[i * n + j]);
        printf("\n");
    }

}

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int N = 0;
    scanf("%d", &N);

    int t; //number of test cases
    scanf("%d", &t);
    while (t--)
    {
        int m, n;
        scanf("%d %d", &m, &n);
        size_t size = m * n * sizeof(float);
        LOG printf("[Matrix Add of two matrices ]\n");

        // Allocate the host input vector A
        float* h_A = (float*)malloc(size);

        // Allocate the host input vector B
        float* h_B = (float*)malloc(size);

        // Allocate the host output vector C
        float* h_C = (float*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the host input vectors

        for (int i = 0; i < n * m; ++i)
        {
            scanf("%f", &h_A[i]);
            scanf("%f", &h_B[i]);

        }


        // Allocate the device input vector A
        float* d_A = NULL;
        err = hipMalloc((void**)&d_A, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Allocate the device input vector B
        float* d_B = NULL;
        err = hipMalloc((void**)&d_B, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Allocate the device output vector C
        float* d_C = NULL;
        err = hipMalloc((void**)&d_C, size);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector A from host to device (error code %s)! \n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)! \n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        // initialize blocksPerGrid and threads Per Block
        int threadsPerBlock = 256;
        int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads \n", threadsPerBlock, blocksPerGrid);
        vecAdd <<blocksPerGrid, threadsPerBlock >>(d_A, d_B, d_C, n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        matrix_add << blocksPerGrid, threadsPerBlock >>  (d_A, d_B, d_C, m, n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        printf("Copy output data from the output device to the host memory\n", hipGetErrorString(err));
        err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector C from device to host(error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        // Verify that the result vector is correct
        for (int i = 0; i < n * m; ++i)
        {
            if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        LOG printf("Test PASSED\n");

        // Free device global memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        // Free host memory
        free(d_A);
        free(d_B);
        free(d_C);

        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        print_matrix(h_C, m, n);

        LOG printf("Done\n");
    }
    return 0;
}
